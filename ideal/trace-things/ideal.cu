
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <unistd.h>
#include <stdio.h>
#include <assert.h>
#include <sys/time.h>
#include <time.h>
#include <stdlib.h>
#include <sys/mman.h>
static void HandleError( hipError_t err, const char *file, int line ) {
    
    if (err != hipSuccess) {
        
        printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    
    }

}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

#define HANDLE_NULL( a ) {if (a == NULL) { \
                            printf( "Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}

inline double diff_s(struct timeval start, struct timeval end)
{
   return ((double) (end.tv_usec - start.tv_usec) / 1000000 + (double) (end.tv_sec - start.tv_sec));
}

__global__ void kernel(uint64_t *in, uint64_t *out)
{
   int tid = threadIdx.x + blockIdx.x*blockDim.x;
   //in = (uint64_t *)malloc(sizeof(uint64_t));
   //out = (uint64_t *)malloc(sizeof(uint64_t));
   out[tid] = in[tid]+tid;
}
__global__ void nullKernel(void)
{

}
void verify(uint64_t *in, uint64_t *out, int numBytes)
{
   int error = 0;
   for(int i =0; i<numBytes; i++)
      if(out[i]!=in[i]+i)
         error = 1;
   if(error)
      printf("ERROR in verification!\n");
   else
      printf("SUCCESS!\n");

}
void cpu_compute(uint64_t *in, uint64_t *out, int numBytes)
{
   for(int i =0; i<numBytes; i++)
      out[i]=in[i]+i;

}

int main( int argc, char *argv[] )
{
    uint64_t *in, *out, *in_d, *out_d;
    int numBytes = 1;
    int opt;
    int benchmarkType = 0;

    while ((opt = getopt(argc, argv, "m:b:")) != -1) {
       switch (opt) {
          case 'm':
             numBytes = atoi(optarg);
             //assert(numBytes%16 == 0 && numBytes<=1024);
             break;
          case 'b':
             benchmarkType = atoi(optarg);
             break;

          default: /* '?' */
             break;
       }
    }


    int num_of_blocks=1;
    int num_of_threads_per_block=numBytes;
    if(numBytes>1024){
       num_of_blocks = 1024;
       num_of_threads_per_block = numBytes/1024;
    }

    HANDLE_ERROR(hipFree(0));
    //cudaHostAlloc(&in,numBytes*sizeof(uint64_t),0);
    //cudaHostAlloc(&out,numBytes*sizeof(uint64_t),0);

    //printf("hostalloc %p\n",in);
    //printf("hostalloc %p\n",out);
    
    //cudaMallocManaged(&in_d,numBytes*sizeof(uint64_t));
    //cudaMallocManaged(&out_d,numBytes*sizeof(uint64_t));

    //printf("managed %p\n",in_d);
    //printf("managed %p\n",out_d);


    printf("Press enter to continue...\n");
    getchar();

    hipHostAlloc(&in,numBytes*sizeof(uint64_t),0);
    hipHostAlloc(&out,numBytes*sizeof(uint64_t),0);
   
    printf("Press enter to continue...\n");
    getchar();

    hipMallocManaged(&in_d,numBytes*sizeof(uint64_t));
    hipMallocManaged(&out_d,numBytes*sizeof(uint64_t));

    printf("Press enter to continue...\n");
    getchar();

    kernel<<<num_of_blocks,num_of_threads_per_block>>>(in,out);

    printf("Press enter to continue...\n");
    getchar();

    hipDeviceSynchronize();
    cpu_compute(in,out,numBytes);

    printf("Press enter to continue...\n");
    getchar();
    hipHostFree(in);
    hipHostFree(out);
    
    printf("Press enter to continue...\n");
    getchar();
    hipFree(in_d);
    hipFree(out_d);

    printf("Press enter to continue...\n");
    getchar();
   
    return 0;
}
