
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <stdio.h>
#include <assert.h>
#include <sys/time.h>
#include <time.h>

static void HandleError( hipError_t err, const char *file, int line ) {
    
    if (err != hipSuccess) {
        
        printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    
    }

}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

#define HANDLE_NULL( a ) {if (a == NULL) { \
                            printf( "Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}

inline double diff_s(struct timeval start, struct timeval end)
{
   return ((double) (end.tv_usec - start.tv_usec) / 1000000 + (double) (end.tv_sec - start.tv_sec));
}

__global__ void readKernel(unsigned char *memory, unsigned char *memoryToRead)
{
   int tid = threadIdx.x + blockIdx.x*blockDim.x;
  // for(int i=0; i<N; i++)
      memory[tid]=memoryToRead[tid];
}
__global__ void writeKernel(unsigned char *memory)
{
   int tid = threadIdx.x + blockIdx.x*blockDim.x;
  // for(int i=0; i<N; i++)
      memory[tid]=5;
}
__global__ void nullKernel(int *memory)
{

}

void verify(unsigned char* memory, int N)
{
   int error = 0;
   for(int i =0; i<N; i++){
      if(memory[i]!=5){
         error = 1;
         break;
      }
   }
   if(error)
      printf("error in verification\n");
   else
      printf("verified SUCCESS\n");
}

int
main( int argc, char *argv[] )
{
    unsigned char *hostAllocd, *cudaMallocd, *cpuMallocd;
    int ITERATIONS = 100000;
    int numBytes = 1;
    struct timeval  tv1, tv2;
    int opt;
    int read=0; //read benchmark? or write?
    int benchmarkType = 0;

      while ((opt = getopt(argc, argv, "m:b:i:r:")) != -1) {
      switch (opt) {
         case 'm':
            numBytes = atoi(optarg);
            assert(numBytes%16 == 0 && numBytes<=1024);
            break;
         case 'b':
            benchmarkType = atoi(optarg);
            break;
         case 'i':
            ITERATIONS = atoi(optarg);
            break;
         case 'r':
            read = atoi(optarg);
            break;
 
         default: /* '?' */
            break;
      }
   }


   cpuMallocd = (unsigned char *)malloc(sizeof(unsigned char)*numBytes);
   assert(cpuMallocd);
   HANDLE_ERROR( hipHostAlloc( &hostAllocd, sizeof(unsigned char)*numBytes, 0 ) );
   for(int k=0;k< numBytes ;k++){
      cpuMallocd[k]=1;
      hostAllocd[k]=1;
   }

   HANDLE_ERROR( hipMalloc( &cudaMallocd, sizeof(unsigned char)*numBytes) );
   HANDLE_ERROR( hipMemcpy( cudaMallocd,hostAllocd, sizeof(unsigned char)*numBytes,hipMemcpyDefault) );

   int num_of_blocks = 1;
   int num_of_threads_per_block = 1;

   //HANDLE_ERROR(cudaDeviceReset());  //this causes kernel launch failure!! check with cuda-memcheck
   HANDLE_ERROR(hipFree(0));
   switch (benchmarkType) {
      case 0: {//read/Write to hostAlloc'd data
                 if(read)
                 {

                    unsigned char *memoryToRead;
                    HANDLE_ERROR( hipHostAlloc( &memoryToRead, sizeof(unsigned char)*numBytes, 0 ) );
                    for(int k=0;k< numBytes ;k++)
                       memoryToRead[k]=5;
                    gettimeofday(&tv1, NULL);
                    for(int i = 0; i < ITERATIONS; i++) {
                       readKernel<<<num_of_blocks,num_of_threads_per_block>>>(hostAllocd,memoryToRead);
                       HANDLE_ERROR( hipDeviceSynchronize());
                    }
                    gettimeofday(&tv2, NULL);
                    hipHostFree(memoryToRead);
                    verify(hostAllocd,numBytes);
                 }
                 else
                 {
                    gettimeofday(&tv1, NULL);
                    for(int i = 0; i < ITERATIONS; i++) {
                       writeKernel<<<num_of_blocks,num_of_threads_per_block>>>(hostAllocd);
                       HANDLE_ERROR( hipDeviceSynchronize());
                    }
                    gettimeofday(&tv2, NULL);
                    verify(hostAllocd,numBytes);
                 }
                 HANDLE_ERROR( hipGetLastError());
                 double elapsedTimeSeconds = diff_s(tv1,tv2);
                 printf("[%s] Latency including kernel launch overhead = %f us\n",(read==1)?"read":"write",elapsedTimeSeconds*1e6/(float)ITERATIONS);
                 break;
              }
   }

   free(cpuMallocd);
   hipFree(cudaMallocd);
   hipHostFree(hostAllocd);
   return 0;
}
