
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <stdio.h>
#include <assert.h>
#include <sys/time.h>
#include <time.h>

static void HandleError( hipError_t err, const char *file, int line ) {
    
    if (err != hipSuccess) {
        
        printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    
    }

}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

#define HANDLE_NULL( a ) {if (a == NULL) { \
                            printf( "Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}

inline double diff_s(struct timeval start, struct timeval end)
{
   return ((double) (end.tv_usec - start.tv_usec) / 1000000 + (double) (end.tv_sec - start.tv_sec));
}

__global__ void readKernel(int *memory, int *memoryToRead)
{
   int tid = threadIdx.x + blockIdx.x*blockDim.x;
   //memory[tid]=memoryToRead[tid];
   //__shared__ int temp; 
   int temp = memoryToRead[tid];
   if(!temp)
      __syncthreads();
}
__global__ void writeKernel(int *memory)
{
   int tid = threadIdx.x + blockIdx.x*blockDim.x;
   memory[tid]=5;
   //memory[tid]++;
}
__global__ void nullKernel(int *memory)
{

}
__global__ void initCudaMallocd(int *memory, int N)
{
   int tid =threadIdx.x;
   if(tid==0){
      for(int k=0;k< N*N/(16*16) ;k++)
         memory[k]=5;
   }
}

void verify(int* memory, int N)
{
   int error = 0;
   for(int i =0; i<N*N/(16*16); i++){
      if(memory[i]!=5){
         error = 1;
         break;
      }
   }
   if(error)
      printf("error in verification\n");
   else
      printf("verified SUCCESS\n");
}
__global__ void verifyCudaMallocd(int* memory, int N)
{
   int tid=threadIdx.x;
   if(tid==0) {
      int error = 0;
      for(int i =0; i<N*N/(16*16); i++){
         if(memory[i]!=5){
            error = 1;
            break;
         }
      }
      if(error)
         printf("error in verification\n");
      else
         printf("verified SUCCESS\n");
   }
}


   int
main( int argc, char *argv[] )
{
    int *hostAllocd, *cudaMallocd, *cpuMallocd;
    int ITERATIONS = 100000;
    int numBytes = 1024;
    struct timeval  tv1, tv2;
    int opt;
    int read=0; //read benchmark? or write?
    int benchmarkType = 0;

      while ((opt = getopt(argc, argv, "m:b:i:r:")) != -1) {
      switch (opt) {
         case 'm':
            numBytes = atoi(optarg);
            //assert(numBytes%16 == 0 && numBytes<=1024);
            break;
         case 'b':
            benchmarkType = atoi(optarg);
            break;
         case 'i':
            ITERATIONS = atoi(optarg);
            break;
         case 'r':
            read = atoi(optarg);
            break;
 
         default: /* '?' */
            break;
      }
   }


   cpuMallocd = (int *)malloc(sizeof(int)*numBytes*numBytes/(16*16));
   assert(cpuMallocd);
   HANDLE_ERROR( hipHostAlloc( &hostAllocd, sizeof(int)*numBytes*numBytes/(16*16), 0 ) );
   for(int k=0;k< numBytes ;k++){
      cpuMallocd[k]=1;
      hostAllocd[k]=1;
   }

   HANDLE_ERROR( hipMalloc( &cudaMallocd, sizeof(int)*numBytes) );
   HANDLE_ERROR( hipMemcpy( cudaMallocd,hostAllocd, sizeof(int)*numBytes,hipMemcpyDefault) );

   //int num_of_blocks=1;
   //int num_of_threads_per_block=numBytes;
   //if(numBytes>1024){
   //   num_of_blocks = 16;
   //   num_of_threads_per_block = numBytes/16;
   //}
   int num_of_blocks,num_of_threads_per_block;
  // if(numBytes==1)
  // {
  //    num_of_blocks=1;
  //    num_of_threads_per_block=1;
  // }
  // else
  // {
  //    assert(numBytes%2==0);
  //    num_of_blocks=2;
  //    num_of_threads_per_block=numBytes/2;
  // }
   num_of_blocks=numBytes/16;
   num_of_threads_per_block=numBytes/16;
   //HANDLE_ERROR(cudaDeviceReset());  //this causes kernel launch failure!! check with cuda-memcheck
   HANDLE_ERROR(hipFree(0));
   switch (benchmarkType) {
      case 0: {//read/Write to hostAlloc'd data
                 if(read)
                 {

                    int *memoryToRead;
                    HANDLE_ERROR( hipHostAlloc( &memoryToRead, sizeof(int)*numBytes*numBytes/(16*16), 0 ) );
                    for(int k=0;k< numBytes*numBytes/(16*16) ;k++)
                       memoryToRead[k]=5;
                    gettimeofday(&tv1, NULL);
                    for(int i = 0; i < ITERATIONS; i++) {
                       readKernel<<<num_of_blocks,num_of_threads_per_block>>>(hostAllocd,memoryToRead);
                       HANDLE_ERROR( hipDeviceSynchronize());
                    }
                    gettimeofday(&tv2, NULL);
                    hipHostFree(memoryToRead);
                    //verify(hostAllocd,numBytes);
                 }
                 else
                 {
                    gettimeofday(&tv1, NULL);
                    for(int i = 0; i < ITERATIONS; i++) {
                       writeKernel<<<num_of_blocks,num_of_threads_per_block>>>(hostAllocd);
                       HANDLE_ERROR( hipDeviceSynchronize());
                    }
                    gettimeofday(&tv2, NULL);
                    //verify(hostAllocd,numBytes);
                 }
                 HANDLE_ERROR( hipGetLastError());
                 double elapsedTimeSeconds = diff_s(tv1,tv2);
                 printf("[%s] Latency including kernel launch overhead = %f us\n",(read==1)?"read":"write",elapsedTimeSeconds*1e6/(float)ITERATIONS);
                 break;
              }
   
      case 1: {//read/Write to cudaMalloc'd data
                 if(read)
                 {

                    int *memoryToRead;
                    HANDLE_ERROR( hipMalloc( &memoryToRead, sizeof(int)*numBytes*numBytes/(16*16) ) );
                    initCudaMallocd<<<1,1>>>(memoryToRead,numBytes);
                    HANDLE_ERROR( hipDeviceSynchronize());
                    gettimeofday(&tv1, NULL);
                    for(int i = 0; i < ITERATIONS; i++) {
                       readKernel<<<num_of_blocks,num_of_threads_per_block>>>(cudaMallocd,memoryToRead);
                       HANDLE_ERROR( hipDeviceSynchronize());
                    }
                    gettimeofday(&tv2, NULL);
                    hipFree(memoryToRead);
                    //verifyCudaMallocd<<<1,1>>>(cudaMallocd,numBytes);
                    //HANDLE_ERROR( cudaDeviceSynchronize());
                 }
                 else
                 {
                    gettimeofday(&tv1, NULL);
                    for(int i = 0; i < ITERATIONS; i++) {
                       writeKernel<<<num_of_blocks,num_of_threads_per_block>>>(cudaMallocd);
                       HANDLE_ERROR( hipDeviceSynchronize());
                    }
                    gettimeofday(&tv2, NULL);
                    //verifyCudaMallocd<<<1,1>>>(cudaMallocd,numBytes);
                    //HANDLE_ERROR( cudaDeviceSynchronize());
                 }
                 HANDLE_ERROR( hipGetLastError());
                 double elapsedTimeSeconds = diff_s(tv1,tv2);
                 printf("[%s] Latency including kernel launch overhead = %f us\n",(read==1)?"read":"write",elapsedTimeSeconds*1e6/(float)ITERATIONS);
                 break;
              }

      case 2:
              {
                 gettimeofday(&tv1, NULL);
                 for(int i = 0; i < ITERATIONS; i++) {
                    nullKernel<<<num_of_blocks,num_of_threads_per_block>>>(0);
                    HANDLE_ERROR( hipDeviceSynchronize());
                 }
                 gettimeofday(&tv2, NULL);
                 HANDLE_ERROR( hipGetLastError());
                 double elapsedTimeSeconds = diff_s(tv1,tv2);
                 printf("null kernel launch overhead = %f us\n",elapsedTimeSeconds*1e6/(float)ITERATIONS);
              
              }
      case 3: {//read/Write to cpu mallocd data
                 if(read)
                 {

                    int temp;
                    int *memoryToRead = (int *)malloc(sizeof(int)*numBytes*numBytes/(16*16) );
                    assert(memoryToRead);
                    for(int k=0;k< numBytes*numBytes/(16*16) ;k++)
                       memoryToRead[k]=5;
                    gettimeofday(&tv1, NULL);
                    for(int i = 0; i < ITERATIONS; i++) {
                       for(int j=0; j<numBytes*numBytes/(16*16); j++){
                          temp=memoryToRead[j];
                          if(!temp)
                             cpuMallocd[j]=temp;
                       }
                    }
                    gettimeofday(&tv2, NULL);
                    free(memoryToRead);
                    //verify(cpuMallocd,numBytes);
                 }
                 else
                 {
                    gettimeofday(&tv1, NULL);
                    for(int i = 0; i < ITERATIONS; i++) {
                       for(int k=0;k< numBytes*numBytes/(16*16) ;k++)
                          cpuMallocd[k]=5;
                    }
                    gettimeofday(&tv2, NULL);
                    verify(cpuMallocd,numBytes);
                 }
                 double elapsedTimeSeconds = diff_s(tv1,tv2);
                 printf("[%s] Latency including kernel launch overhead = %f us\n",(read==1)?"read":"write",elapsedTimeSeconds*1e6/(float)ITERATIONS);
                 break;
              }
   

   }

   free(cpuMallocd);
   hipFree(cudaMallocd);
   hipHostFree(hostAllocd);
   hipDeviceReset();
   return 0;
}
