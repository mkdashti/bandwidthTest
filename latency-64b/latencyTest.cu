
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <unistd.h>
#include <stdio.h>
#include <assert.h>
#include <sys/time.h>
#include <time.h>
#include <stdlib.h>
static void HandleError( hipError_t err, const char *file, int line ) {
    
    if (err != hipSuccess) {
        
        printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    
    }

}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

#define HANDLE_NULL( a ) {if (a == NULL) { \
                            printf( "Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}

inline double diff_s(struct timeval start, struct timeval end)
{
   return ((double) (end.tv_usec - start.tv_usec) / 1000000 + (double) (end.tv_sec - start.tv_sec));
}

__global__ void readKernel(uint64_t *memory, uint64_t *memoryToRead)
{
   int tid = threadIdx.x + blockIdx.x*blockDim.x;
   //memory[tid]=memoryToRead[tid];
   //__shared__ uint64_t temp; 
   uint64_t temp = memoryToRead[tid];
   if(!temp)
      __syncthreads();
}
__global__ void writeKernel(uint64_t *memory)
{
   int tid = threadIdx.x + blockIdx.x*blockDim.x;
   memory[tid]=5;
}
__global__ void nullKernel(int *memory)
{

}
__global__ void initCudaMallocd(uint64_t *memory, int N)
{
   int tid =threadIdx.x;
   if(tid==0){
      for(int k=0;k< N ;k++)
         memory[k]=5;
   }
}

void verify(uint64_t* memory, int N)
{
   int error = 0;
   for(int i =0; i<N; i++){
      if(memory[i]!=5){
         error = 1;
         break;
      }
   }
   if(error)
      printf("error in verification\n");
   else
      printf("verified SUCCESS\n");
}
__global__ void verifyCudaMallocd(uint64_t* memory, int N)
{
   int tid=threadIdx.x;
   if(tid==0) {
      int error = 0;
      for(int i =0; i<N; i++){
         if(memory[i]!=5){
            error = 1;
            break;
         }
      }
      if(error)
         printf("error in verification\n");
      else
         printf("verified SUCCESS\n");
   }
}


   int
main( int argc, char *argv[] )
{
    uint64_t *hostAllocd, *cudaMallocd, *cpuMallocd;
    int ITERATIONS = 100000;
    int numBytes = 1;
    struct timeval  tv1, tv2;
    int opt;
    int read=0; //read benchmark? or write?
    int benchmarkType = 0;

      while ((opt = getopt(argc, argv, "m:b:i:r:")) != -1) {
      switch (opt) {
         case 'm':
            numBytes = atoi(optarg);
            //assert(numBytes%16 == 0 && numBytes<=1024);
            break;
         case 'b':
            benchmarkType = atoi(optarg);
            break;
         case 'i':
            ITERATIONS = atoi(optarg);
            break;
         case 'r':
            read = atoi(optarg);
            break;
 
         default: /* '?' */
            break;
      }
   }


   cpuMallocd = (uint64_t *)malloc(sizeof(uint64_t)*numBytes);
   assert(cpuMallocd);
   HANDLE_ERROR( hipHostAlloc( &hostAllocd, sizeof(uint64_t)*numBytes, 0 ) );
   for(int k=0;k< numBytes ;k++){
      cpuMallocd[k]=1;
      hostAllocd[k]=1;
   }

   HANDLE_ERROR( hipMalloc( &cudaMallocd, sizeof(uint64_t)*numBytes) );
   HANDLE_ERROR( hipMemcpy( cudaMallocd,hostAllocd, sizeof(uint64_t)*numBytes,hipMemcpyDefault) );

   int num_of_blocks=1;
   int num_of_threads_per_block=numBytes;
   if(numBytes>1024){
      num_of_blocks = 16;
      num_of_threads_per_block = numBytes/16;
   }

   //HANDLE_ERROR(cudaDeviceReset());  //this causes kernel launch failure!! check with cuda-memcheck
   HANDLE_ERROR(hipFree(0));
   switch (benchmarkType) {
      case 0: {//read/Write to hostAlloc'd data
                 if(read)
                 {

                    uint64_t *memoryToRead;
                    HANDLE_ERROR( hipHostAlloc( &memoryToRead, sizeof(uint64_t)*numBytes, 0 ) );
                    for(int k=0;k< numBytes ;k++)
                       memoryToRead[k]=5;
                    gettimeofday(&tv1, NULL);
                    for(int i = 0; i < ITERATIONS; i++) {
                       readKernel<<<num_of_blocks,num_of_threads_per_block>>>(hostAllocd,memoryToRead);
                       HANDLE_ERROR( hipDeviceSynchronize());
                    }
                    gettimeofday(&tv2, NULL);
                    hipHostFree(memoryToRead);
                    //verify(hostAllocd,numBytes);
                 }
                 else
                 {
                    gettimeofday(&tv1, NULL);
                    for(int i = 0; i < ITERATIONS; i++) {
                       writeKernel<<<num_of_blocks,num_of_threads_per_block>>>(hostAllocd);
                       HANDLE_ERROR( hipDeviceSynchronize());
                    }
                    gettimeofday(&tv2, NULL);
                    verify(hostAllocd,numBytes);
                 }
                 HANDLE_ERROR( hipGetLastError());
                 double elapsedTimeSeconds = diff_s(tv1,tv2);
                 printf("HostAlloc [%s] Latency including kernel launch overhead = %f us\n",(read==1)?"read":"write",elapsedTimeSeconds*1e6/(float)ITERATIONS);
                 break;
              }
   
      case 1: {//read/Write to cudaMalloc'd data
                 if(read)
                 {

                    uint64_t *memoryToRead;
                    HANDLE_ERROR( hipMalloc( &memoryToRead, sizeof(uint64_t)*numBytes ) );
                    initCudaMallocd<<<1,1>>>(memoryToRead,numBytes);
                    HANDLE_ERROR( hipDeviceSynchronize());
                    gettimeofday(&tv1, NULL);
                    for(int i = 0; i < ITERATIONS; i++) {
                       readKernel<<<num_of_blocks,num_of_threads_per_block>>>(cudaMallocd,memoryToRead);
                       HANDLE_ERROR( hipDeviceSynchronize());
                    }
                    gettimeofday(&tv2, NULL);
                    hipFree(memoryToRead);
                    //verifyCudaMallocd<<<1,1>>>(cudaMallocd,numBytes);
                    //HANDLE_ERROR( cudaDeviceSynchronize());
                 }
                 else
                 {
                    gettimeofday(&tv1, NULL);
                    for(int i = 0; i < ITERATIONS; i++) {
                       writeKernel<<<num_of_blocks,num_of_threads_per_block>>>(cudaMallocd);
                       HANDLE_ERROR( hipDeviceSynchronize());
                    }
                    gettimeofday(&tv2, NULL);
                    //verifyCudaMallocd<<<1,1>>>(cudaMallocd,numBytes);
                    //HANDLE_ERROR( cudaDeviceSynchronize());
                 }
                 HANDLE_ERROR( hipGetLastError());
                 double elapsedTimeSeconds = diff_s(tv1,tv2);
                 printf("CudaMalloc [%s] Latency including kernel launch overhead = %f us\n",(read==1)?"read":"write",elapsedTimeSeconds*1e6/(float)ITERATIONS);
                 break;
              }

      case 2:
              {
                 gettimeofday(&tv1, NULL);
                 for(int i = 0; i < ITERATIONS; i++) {
                    nullKernel<<<num_of_blocks,num_of_threads_per_block>>>(0);
                    HANDLE_ERROR( hipDeviceSynchronize());
                 }
                 gettimeofday(&tv2, NULL);
                 HANDLE_ERROR( hipGetLastError());
                 double elapsedTimeSeconds = diff_s(tv1,tv2);
                 printf("null kernel launch overhead = %f us\n",elapsedTimeSeconds*1e6/(float)ITERATIONS);
                 break;
              
              }
      case 3: {//read/Write to cpu mallocd data
                 uint64_t *memory_to_access = (uint64_t *)malloc(sizeof(uint64_t)*numBytes );
                 assert(memory_to_access);
                 if(read)
                 {
                    for(int k=0;k< numBytes ;k++)
                       memory_to_access[k]=5;

                    uint64_t fake;
                    gettimeofday(&tv1, NULL);
                    for(int i=0; i<ITERATIONS; i++) {
                       for (int j = 0; j < (numBytes); j += 8) {
                          fake += memory_to_access[j];
                          fake += memory_to_access[j + 1];
                          fake += memory_to_access[j + 2];
                          fake += memory_to_access[j + 3];
                          fake += memory_to_access[j + 4];
                          fake += memory_to_access[j + 5];
                          fake += memory_to_access[j + 6];
                          fake += memory_to_access[j + 7];
                       }
                    }
                    gettimeofday(&tv2, NULL);
                 }
                 else
                 {
                    uint64_t fake=5;
                    gettimeofday(&tv1, NULL);
                    for(int i=0; i<ITERATIONS; i++) {
                       for (int j = 0; j < (numBytes); j += 8) {
                          memory_to_access[j] = fake;
                          memory_to_access[j + 1] = fake;
                          memory_to_access[j + 2] = fake;
                          memory_to_access[j + 3] = fake;
                          memory_to_access[j + 4] = fake;
                          memory_to_access[j + 5] = fake;
                          memory_to_access[j + 6] = fake;
                          memory_to_access[j + 7] = fake;
                       }
                    }
                    gettimeofday(&tv2, NULL);
                 }
                 double elapsedTimeSeconds = diff_s(tv1,tv2);
                 printf("cpu malloc [%s] Latency = %f us\n",(read==1)?"read":"write",elapsedTimeSeconds*1e6/(float)ITERATIONS);
                 free(memory_to_access);
                 break;
              }
      case 4: {//read/Write to cpu but hostsllocd data
                 uint64_t *memory_to_access;
                 HANDLE_ERROR(hipHostAlloc(&memory_to_access,sizeof(uint64_t)*numBytes,0));
                 if(read)
                 {
                    for(int k=0;k< numBytes ;k++)
                       memory_to_access[k]=5;

                    uint64_t fake;
                    gettimeofday(&tv1, NULL);
                    for(int i=0; i<ITERATIONS; i++) {
                       for (int j = 0; j < (numBytes); j += 8) {
                          fake += memory_to_access[j];
                          fake += memory_to_access[j + 1];
                          fake += memory_to_access[j + 2];
                          fake += memory_to_access[j + 3];
                          fake += memory_to_access[j + 4];
                          fake += memory_to_access[j + 5];
                          fake += memory_to_access[j + 6];
                          fake += memory_to_access[j + 7];
                       }
                    }
                    gettimeofday(&tv2, NULL);
                 }
                 else
                 {
                    uint64_t fake=5;
                    gettimeofday(&tv1, NULL);
                    for(int i=0; i<ITERATIONS; i++) {
                       for (int j = 0; j < (numBytes); j += 8) {
                          memory_to_access[j] = fake;
                          memory_to_access[j + 1] = fake;
                          memory_to_access[j + 2] = fake;
                          memory_to_access[j + 3] = fake;
                          memory_to_access[j + 4] = fake;
                          memory_to_access[j + 5] = fake;
                          memory_to_access[j + 6] = fake;
                          memory_to_access[j + 7] = fake;
                       }
                    }
                    gettimeofday(&tv2, NULL);
                 }
                 double elapsedTimeSeconds = diff_s(tv1,tv2);
                 printf("cpu malloc [%s] Latency = %f us\n",(read==1)?"read":"write",elapsedTimeSeconds*1e6/(float)ITERATIONS);
                 hipHostFree(memory_to_access);
                 break;
              }
       
   }

   free(cpuMallocd);
   hipFree(cudaMallocd);
   hipHostFree(hostAllocd);
   hipDeviceReset();
   return 0;
}