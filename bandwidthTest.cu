#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <sched.h>
#include <sys/sysinfo.h>
#include <pthread.h>
#include <sched.h>
#include <sys/syscall.h> 

#include <sys/time.h>
#include <time.h>
#include <unistd.h>

static void HandleError( hipError_t err, const char *file, int line ) {
    
    if (err != hipSuccess) {
        
        printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    
    }

}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

#define HANDLE_NULL( a ) {if (a == NULL) { \
                            printf( "Host memory failed in %s at line %d\n", \
                                    __FILE__, __LINE__ ); \
                            exit( EXIT_FAILURE );}}

inline double diff_s(struct timeval start, struct timeval end)
{
   return ((double) (end.tv_usec - start.tv_usec) / 1000000 + (double) (end.tv_sec - start.tv_sec));
}

__device__ void busywait(int cycles)
{
   clock_t current_time;
   current_time = clock64();
   int until = current_time + cycles;
   while (until > current_time) {
      current_time = clock64();
   }
}

__global__ void copyKernel(unsigned char *output, unsigned char *input, int N) { 
   int tid = threadIdx.x + blockIdx.x*blockDim.x;
   if(tid < N)
      output[tid] = input[tid];

} 
__global__ void nullKernel(unsigned char *output, unsigned char *input, int N) { 

} 


static pid_t gettid(void) {                                                                      
   return syscall(__NR_gettid);
}

void set_affinity(int tid, int core_id) {
   cpu_set_t mask;
   CPU_ZERO(&mask);
   CPU_SET(core_id, &mask);

   int r = sched_setaffinity(tid, sizeof(mask), &mask);                                                                  
   if (r < 0) {
      fprintf(stderr, "couldn't set affinity for %d\n", core_id);
      exit(1);
   }
}


typedef struct {                                                                                                        
   int id;
   unsigned char *input;
   unsigned char *output;
   int memSize;
} parm;

void *work( void *arg ) {
   parm *p=(parm *)arg;
   int tid = gettid();                                                                                                   
   set_affinity(tid, (p->id)%get_nprocs());

   memcpy(p->output,p->input,p->memSize);
   return 0;
}
void launch_cpu_threads(int nthreads, unsigned char **out, unsigned char **in, int memSize)
{
   pthread_t *threads;
   pthread_attr_t attr;
   parm *p;
   int j;
   
   threads=(pthread_t *)malloc(nthreads * sizeof(pthread_t));
   if(threads == NULL) {
      printf("ERROR malloc failed to create CPU threads\n");
      exit(1);
   }
   pthread_attr_init(&attr);
   p=(parm *)malloc(nthreads * sizeof(parm));

   for (j=0; j<nthreads; j++)
   {
      p[j].id=j;
      p[j].input=in[j];
      p[j].output=out[j];
      p[j].memSize=memSize;
      if(pthread_create(&threads[j], &attr, work, (void *)(p+j))!=0)
      {
         printf("ERROR creating threads\n");
         exit(1);
      }
   }

   for (j=0; j<nthreads; j++)
   {
      if(pthread_join(threads[j],NULL)!=0) {
         printf("ERROR in joing threads\n");
         exit(1);
      }
   }

   pthread_attr_destroy(&attr);
   free(p);
}


int main(int argc, char *argv[]) 
{
   struct timeval  tv1, tv2;

   int opt;

   int num_of_blocks = 1024;
	int num_of_threads_per_block = 1024;
   int memSize = 4*1024*1024;
   int benchmarkType = 0;
   int ITERATIONS = 10;
   int t = 512;
 
   while ((opt = getopt(argc, argv, "m:b:i:t:")) != -1) {
      switch (opt) {
         case 'm':
            memSize = atoi(optarg)*1024*1024;
            break;
         case 'b':
            benchmarkType = atoi(optarg);
            break;
         case 'i':
            ITERATIONS = atoi(optarg);
            break;
         case 't':
            t = atoi(optarg);
            break;
 
         default: /* '?' */
            break;
      }
   }
   num_of_blocks = memSize/t;
   num_of_threads_per_block = t;
   assert(num_of_blocks <= 2147483647);
   assert(num_of_threads_per_block <= 1024);
   int N = num_of_blocks * num_of_threads_per_block;
   HANDLE_ERROR(hipDeviceReset());
   hipFree(0); //set context so that overhead won't be later accounted

   unsigned char *cpuMemory,*inputhostallocMemory,*outputhostallocMemory,*inputcudamallocMemory,*outputcudamallocMemory,
                 *inputmanagedMemory, *outputmanagedMemory;
   cpuMemory = (unsigned char *)malloc(memSize);
   assert(cpuMemory);
   for(int i = 0; i < memSize/sizeof(unsigned char); i++)
      cpuMemory[i] = (unsigned char)(i & 0xff);

   HANDLE_ERROR( hipHostAlloc( (void**)& inputhostallocMemory, sizeof(unsigned char)*memSize, hipHostMallocDefault) );
   HANDLE_ERROR( hipHostAlloc( (void**)& outputhostallocMemory, sizeof(unsigned char)*memSize, hipHostMallocDefault) );
   HANDLE_ERROR( hipMalloc( (void**)& inputcudamallocMemory, sizeof(unsigned char)*memSize) );
   HANDLE_ERROR( hipMalloc( (void**)& outputcudamallocMemory, sizeof(unsigned char)*memSize) );
   HANDLE_ERROR( hipMemcpy(inputcudamallocMemory,cpuMemory, sizeof(unsigned char)*memSize,hipMemcpyDefault) );
   HANDLE_ERROR( hipMemcpy(inputhostallocMemory,cpuMemory, sizeof(unsigned char)*memSize,hipMemcpyDefault) );

  
   switch (benchmarkType) {
      case 0: {//Device to Device memcpy test 

                 gettimeofday(&tv1, NULL);
                 for(int i = 0; i < ITERATIONS; i++) {
                    HANDLE_ERROR( hipMemcpy(outputcudamallocMemory,inputcudamallocMemory, sizeof(unsigned char)*memSize,hipMemcpyDefault) );
                 }
                 HANDLE_ERROR( hipDeviceSynchronize());
                 gettimeofday(&tv2, NULL);
                 double elapsedTimeSeconds = diff_s(tv1,tv2);
                 printf("elapsedTime per iteration = %f\n",elapsedTimeSeconds/ITERATIONS);
                 //we multiply by two since the DeviceToDevice copy involves both reading and writing to device memory
                 float bandwidth = 2.0f * ((double)memSize/(1024*1024*1024))*ITERATIONS/elapsedTimeSeconds;
                 //float bandwidth =  2.0f * ((float)(1<<10) * memSize * (float)ITERATIONS) / (elapsedTimeSeconds *(1000.0) * (float)(1 << 20));

                 printf("DeviceToDevice Memcpy Bandwitdh = %f GB/s\n",bandwidth);
                 break;
              }
      case 1: {//custom kernel with cuda malloced memory
                 gettimeofday(&tv1, NULL);
                 for(int i = 0; i < ITERATIONS; i++) {
                    nullKernel<<<num_of_blocks,num_of_threads_per_block>>>(outputcudamallocMemory,inputcudamallocMemory,N);
                 }
                 HANDLE_ERROR( hipDeviceSynchronize());
                 gettimeofday(&tv2, NULL);
                 HANDLE_ERROR( hipGetLastError());
                 double nullElapsedTime = diff_s(tv1,tv2);

                 gettimeofday(&tv1, NULL);
                 for(int i = 0; i < ITERATIONS; i++) {
                    copyKernel<<<num_of_blocks,num_of_threads_per_block>>>(outputcudamallocMemory,inputcudamallocMemory,N);
                 }
                 HANDLE_ERROR( hipDeviceSynchronize());
                 gettimeofday(&tv2, NULL);
                 HANDLE_ERROR( hipGetLastError());
                 double elapsedTimeSeconds = diff_s(tv1,tv2);
                 printf("elapsedTime per iteration = %f\n",elapsedTimeSeconds/ITERATIONS);
                 //we multiply by two since the DeviceToDevice copy involves both reading and writing to device memory
                 float bandwidth = 2.0f * ((double)memSize/(1024*1024*1024))*ITERATIONS/elapsedTimeSeconds;
                 float bandwidth_ex = 2.0f * ((double)memSize/(1024*1024*1024))*ITERATIONS/(elapsedTimeSeconds-nullElapsedTime);
                 //float bandwidth =  2.0f * ((float)(1<<10) * memSize * (float)ITERATIONS) / (elapsedTimeSeconds *(1000.0) * (float)(1 << 20));

                 printf("Custom kernel(hipMalloc) memcpy Bandwitdh including kernel launch overhead = %f GB/s\n",bandwidth);
                 printf("Custom kernel(hipMalloc) memcpy Bandwitdh excluding kernel launch overhead = %f GB/s\n",bandwidth_ex);
                 break;
              }
      case 2: {//Custom kernel with host allocated memory
                 gettimeofday(&tv1, NULL);
                 for(int i = 0; i < ITERATIONS; i++) {
                    nullKernel<<<num_of_blocks,num_of_threads_per_block>>>(outputhostallocMemory,inputhostallocMemory,N);
                 }
                 HANDLE_ERROR( hipDeviceSynchronize());
                 gettimeofday(&tv2, NULL);
                 HANDLE_ERROR( hipGetLastError());
                 double nullElapsedTime = diff_s(tv1,tv2);

 
                 gettimeofday(&tv1, NULL);
                 for(int i = 0; i < ITERATIONS; i++) {
                    copyKernel<<<num_of_blocks,num_of_threads_per_block>>>(outputhostallocMemory,inputhostallocMemory,N);
                 }
                 HANDLE_ERROR( hipDeviceSynchronize());
                 gettimeofday(&tv2, NULL);
                 HANDLE_ERROR( hipGetLastError());
                 double elapsedTimeSeconds = diff_s(tv1,tv2);
                 printf("elapsedTime per iteration = %f\n",elapsedTimeSeconds/ITERATIONS);
                 //we multiply by two since the DeviceToDevice copy involves both reading and writing to device memory
                 float bandwidth = 2.0f * ((double)memSize/(1024*1024*1024))*ITERATIONS/elapsedTimeSeconds;
                 float bandwidth_ex = 2.0f * ((double)memSize/(1024*1024*1024))*ITERATIONS/(elapsedTimeSeconds-nullElapsedTime);
                 //float bandwidth =  2.0f * ((float)(1<<10) * memSize * (float)ITERATIONS) / (elapsedTimeSeconds *(1000.0) * (float)(1 << 20));

                 printf("Custom kernel(hipHostAlloc) memcpy Bandwitdh = %f GB/s\n",bandwidth);
                 printf("Custom kernel(hipHostAlloc) memcpy Bandwitdh excluding kernel launch overhead = %f GB/s\n",bandwidth_ex);
                 break;
              }
      case 3: {//host allocated memory copy test

                 gettimeofday(&tv1, NULL);
                 for(int i = 0; i < ITERATIONS; i++) {
                    HANDLE_ERROR( hipMemcpyAsync(outputhostallocMemory,inputhostallocMemory, sizeof(unsigned char)*memSize,hipMemcpyDefault) );
                 }
                 HANDLE_ERROR( hipDeviceSynchronize());
                 gettimeofday(&tv2, NULL);
                 HANDLE_ERROR( hipGetLastError());
                 double elapsedTimeSeconds = diff_s(tv1,tv2);
                 printf("elapsedTime per iteration = %f\n",elapsedTimeSeconds/ITERATIONS);
                 //we multiply by two since the DeviceToDevice copy involves both reading and writing to device memory
                 float bandwidth = 2.0f * ((double)memSize/(1024*1024*1024))*ITERATIONS/elapsedTimeSeconds;
                 //float bandwidth =  2.0f * ((float)(1<<10) * memSize * (float)ITERATIONS) / (elapsedTimeSeconds *(1000.0) * (float)(1 << 20));

                 printf("Device to Device hipHostAlloc memcpy Bandwitdh = %f GB/s\n",bandwidth);
                 break;
              }

      case 4: {//managed memory copy test
                 HANDLE_ERROR( hipMallocManaged( (void**)& inputmanagedMemory, sizeof(unsigned char)*memSize) );
                 HANDLE_ERROR( hipMallocManaged( (void**)& outputmanagedMemory, sizeof(unsigned char)*memSize) );
                 HANDLE_ERROR( hipMemcpy(inputmanagedMemory,cpuMemory, sizeof(unsigned char)*memSize,hipMemcpyDefault) );
                 gettimeofday(&tv1, NULL);
                 for(int i = 0; i < ITERATIONS; i++) {
                    nullKernel<<<num_of_blocks,num_of_threads_per_block>>>(outputmanagedMemory,inputmanagedMemory,N);
                 }
                 HANDLE_ERROR( hipDeviceSynchronize());
                 gettimeofday(&tv2, NULL);
                 HANDLE_ERROR( hipGetLastError());
                 double nullElapsedTime = diff_s(tv1,tv2);

 

                 gettimeofday(&tv1, NULL);
                 for(int i = 0; i < ITERATIONS; i++) {
                    copyKernel<<<num_of_blocks,num_of_threads_per_block>>>(outputmanagedMemory,inputmanagedMemory,N);
                 }
                 HANDLE_ERROR( hipDeviceSynchronize());
                 gettimeofday(&tv2, NULL);
                 HANDLE_ERROR( hipGetLastError());
                 double elapsedTimeSeconds = diff_s(tv1,tv2);
                 printf("elapsedTime per iteration = %f\n",elapsedTimeSeconds/ITERATIONS);
                 //we multiply by two since the DeviceToDevice copy involves both reading and writing to device memory
                 float bandwidth = 2.0f * ((double)memSize/(1024*1024*1024))*ITERATIONS/elapsedTimeSeconds;
                 float bandwidth_ex = 2.0f * ((double)memSize/(1024*1024*1024))*ITERATIONS/(elapsedTimeSeconds-nullElapsedTime);
                 //float bandwidth =  2.0f * ((float)(1<<10) * memSize * (float)ITERATIONS) / (elapsedTimeSeconds *(1000.0) * (float)(1 << 20));

                 printf("Custom kernel (managed memory)  Bandwitdh = %f GB/s\n",bandwidth);
                 printf("Custom kernel (managed memory)  Bandwitdh excluding kernel launch overhead = %f GB/s\n",bandwidth_ex);
                 hipFree(inputmanagedMemory);
                 hipFree(outputmanagedMemory);

                 break;
              }
      case 5: {//Custom kernel with host allocated to malloc copy
                 gettimeofday(&tv1, NULL);
                 for(int i = 0; i < ITERATIONS; i++) {
                    nullKernel<<<num_of_blocks,num_of_threads_per_block>>>(outputcudamallocMemory,inputhostallocMemory,N);
                 }
                 HANDLE_ERROR( hipDeviceSynchronize());
                 gettimeofday(&tv2, NULL);
                 HANDLE_ERROR( hipGetLastError());
                 double nullElapsedTime = diff_s(tv1,tv2);

 
                 gettimeofday(&tv1, NULL);
                 for(int i = 0; i < ITERATIONS; i++) {
                    copyKernel<<<num_of_blocks,num_of_threads_per_block>>>(outputcudamallocMemory,inputhostallocMemory,N);
                 }
                 HANDLE_ERROR( hipDeviceSynchronize());
                 gettimeofday(&tv2, NULL);
                 HANDLE_ERROR( hipGetLastError());
                 double elapsedTimeSeconds = diff_s(tv1,tv2);
                 printf("elapsedTime per iteration = %f\n",elapsedTimeSeconds/ITERATIONS);
                 //we multiply by two since the DeviceToDevice copy involves both reading and writing to device memory
                 float bandwidth = 2.0f * ((double)memSize/(1024*1024*1024))*ITERATIONS/elapsedTimeSeconds;
                 float bandwidth_ex = 2.0f * ((double)memSize/(1024*1024*1024))*ITERATIONS/(elapsedTimeSeconds-nullElapsedTime);
                 //float bandwidth =  2.0f * ((float)(1<<10) * memSize * (float)ITERATIONS) / (elapsedTimeSeconds *(1000.0) * (float)(1 << 20));

                 printf("Custom kernel(hipHostAlloc to hipMalloc) memcpy Bandwitdh = %f GB/s\n",bandwidth);
                 printf("Custom kernel(hipHostAlloc to hipMalloc) memcpy Bandwitdh excluding kernel launch overhead = %f GB/s\n",bandwidth_ex);
                 break;
              }
      case 6: {//Cpu malloc to malloc
 
                 unsigned char *mallocdOut,*mallocdIn;
                 mallocdOut = (unsigned char *)malloc(sizeof(unsigned char)*memSize);
                 mallocdIn = (unsigned char *)malloc(sizeof(unsigned char)*memSize);
                 for(int i=0; i<memSize/sizeof(unsigned char); i++)
                    mallocdIn[i]=5;
                 if(!mallocdOut || !mallocdIn) {
                    printf("ERROR in malloc\n");
                    return -1;
                 }
                 for(int i=0; i<memSize/sizeof(unsigned char); i++)
                    mallocdIn[i]=5;
                
                 gettimeofday(&tv1, NULL);
                 for(int i = 0; i < ITERATIONS; i++) {
                    memcpy(mallocdOut,mallocdIn,sizeof(unsigned char)*memSize);
                 }
                 gettimeofday(&tv2, NULL);
                 double elapsedTimeSeconds = diff_s(tv1,tv2);
                 printf("elapsedTime per iteration = %f\n",elapsedTimeSeconds/ITERATIONS);
                 //we multiply by two since the DeviceToDevice copy involves both reading and writing to device memory
                 float bandwidth =  ((double)memSize/(1024*1024*1024))*ITERATIONS/elapsedTimeSeconds;

                 printf("cpu malloc to malloc memcpy Bandwitdh = %f GB/s\n",bandwidth);
                 free(mallocdOut);
                 free(mallocdIn);
                 break;
              }
       case 7: {//Cpu multithreaded malloc to malloc

                 const int nthreads=4; 
                 unsigned char *mallocdOut[nthreads],*mallocdIn[nthreads];
                 for(int i=0; i<nthreads; i++) {
                    mallocdOut[i] = (unsigned char *)malloc(sizeof(unsigned char)*memSize);
                    mallocdIn[i] = (unsigned char *)malloc(sizeof(unsigned char)*memSize);
                 }
                  if(!mallocdOut || !mallocdIn) {
                    printf("ERROR in malloc\n");
                    return -1;
                 }
                for(int i=0; i<memSize/sizeof(unsigned char); i++){
                    mallocdIn[0][i]=5;
                    mallocdIn[1][i]=5;
                    mallocdIn[2][i]=5;
                    mallocdIn[3][i]=5;
                }
                 gettimeofday(&tv1, NULL);
                 for(int i = 0; i < ITERATIONS; i++) {
                    launch_cpu_threads(nthreads,mallocdOut,mallocdIn,memSize);
                 }
                 gettimeofday(&tv2, NULL);
                 double elapsedTimeSeconds = diff_s(tv1,tv2);
                 printf("elapsedTime per iteration = %f\n",elapsedTimeSeconds/ITERATIONS);
                 //we multiply by two since the DeviceToDevice copy involves both reading and writing to device memory
                 float bandwidth = 4 * ((double)memSize/(1024*1024*1024))*ITERATIONS/elapsedTimeSeconds;
                 printf("cpu malloc to malloc memcpy Bandwitdh = %f GB/s\n",bandwidth);
                 for(int i=0; i<nthreads; i++){
                    free(mallocdOut[i]);
                    free(mallocdIn[i]);
                 }
                 break;
              }
 





   }
   free(cpuMemory);
   hipHostFree(inputhostallocMemory);
   hipHostFree(outputhostallocMemory);
   hipFree(inputcudamallocMemory);
   hipFree(outputcudamallocMemory);
   return 0; 
}
